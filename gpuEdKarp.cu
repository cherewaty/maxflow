#include "hip/hip_runtime.h"
#include "gpuEdKarp.h"




__device__ int indexFinder(int i,int n, int j){
    return ((i) * (n) + (j));
}

__global__ void backTrack(int *parents,int *flowMatrix, int s,int v,int tempCapacity,int n){
    while (v != s){
        int u = parents[v];
        flowMatrix[indexFinder(u,v,n)] += tempCapacity;
        flowMatrix[indexFinder(v,u,n)] -= tempCapacity;
        v = u;
      }
}

__global__ void nextQueue(int queueSize,int *addedToQueue,bool *hasResult,int *result,int *queue,int *queueIndex int *parents, int *pathCapacities,int u,int n,int *capacities, int *flowMatrix){
    *addedToQueue = 0;
    for (int v = 0; v < n; v++){
        if (u == v){
          continue;
        }
        int residual = capacities[indexFinder(u, v, n)] - flowMatrix[indexFinder(u, v, n)];
        if ((residual > 0) && (parents[v] == -1)){
          parents[v] = u;
          pathCapacities[v] = fminf(pathCapacities[u], residual);
          if (v != t){
            queue[queueSize + *addedToQueue]=v;
            *addedToQueue++;
          }else{
            *hasResult = true;
            *result = pathCapacities[t];
            break;
          }
        }
      }
}

//Code Reference https://github.com/kaletap/bfs-cuda-gpu/blob/master/src/gpu/simple/bfs_simple.cu
int BFS(Graph *g, int *flowMatrix, int *parents, int *pathCapacities, int s, int t,int *d_parents,int *d_flowMaxtrix,int *d_pathCapacities){
  memset(parents, -1, (g->n * sizeof(int)));
  memset(pathCapacities, 0, (g->n * sizeof(int)));
  parents[s] = s;
  pathCapacities[s] = std::numeric_limits<int>::max();

  int currentQueueSize = 1;

  int *queue;
  int queueIndex = 0;
  int addedToQueue = 0;
  int result = 0;
  
  int *d_queue;
  int *d_queueIndex;
  int *d_addedToQueue;
  int *d_result;
  bool *d_hasResult;
  int size = g->n * sizeof(int);

  malloc(queue,size);
  hipMalloc((void **)&d_queue, size);
  hipMalloc((void**)&d_queueIndex,sizeof(int));
  hipMalloc((void**)&d_addedToQueue,sizeof(int));
  hipMalloc((void**)&d_hasResult,sizeof(bool));
  hipMalloc((void**)&d_result,sizeof(int));
  *hasResult = false;
  queue[0] = s;

  while(currentQueueSize > 0){
    int u = queue[queueIndex];
    hipMemcpy(d_hasResult,hasResult,sizeof(bool),hipMemcpyHostToDevice);
    hipMemcpy(d_resutl,result,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_addedToQueue,addedToQueue,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_queue,queue,size,hipMemcpyHostToDevice);
    hipMemcpy(d_queueIndex,queueIndex,sizeof(int),hipMemcpyHostToDevice);

    nextQueue<<<1,1>>>(currentQueueSize,d_addedToQueue,d_hasResult,d_result,d_queue,d_queueIndex,d_parents,d_pathCapacities,u,g->n,d_capacities,d_flowMaxtrix);
    hipMemcpy(hasResult,d_hasResult,sizeof(bool),hipMemcpyDeviceToHost);
    hipMemcpy(result,d_resutl,sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(addedToQueue,d_addedToQueue,sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(parents,d_parents,size,hipMemcpyDeviceToHost);
    hipMemcpy(pathCapacities,d_pathCapacities,size,hipMemcpyDeviceToHost);
    if(hasResult){
        free(queue)
        hipFree(d_hasResult);
        hipFree(d_result);
        hipFree(d_queueIndex);
        hipFree(d_addedToQueue);
        hipFree(d_queue);
        return result
    }else {
        hipMemcpy(queue,d_queue,size,hipMemcpyDeviceToHost);
        currentQueueSize = addedToQueue -1;
        queueIndex++;
    }

  }
  free(queue)
  hipFree(d_hasResult);
  hipFree(d_result);
  hipFree(d_queueIndex);
  hipFree(d_addedToQueue);
  hipFree(d_queue);
  return 0;
}





Flow *edKarpGpu(Graph *g, int s, int t){
    int flow = 0;
    int *flowMatrix = (int *)malloc(g->n * g->n * sizeof(int));
    int *parents = (int *)malloc(g->n * sizeof(int));
    int *pathCapacities = (int *)malloc(g->n* sizeof(int));
    
    
    int *d_flowMaxtrix;
    int *d_parents;
    int *d_pathCapacities;

    hipMalloc((void **)&d_flowMaxtrix,g->n * g->n * sizeof(int));
    hipMalloc((void **)&d_parents,g->n * sizeof(int));
    hipMalloc((void **)&d_pathCapacities,g->n* sizeof(int));

    while (true){
      int tempCapacity = BFS(g, flowMatrix, parents, pathCapacities, s, t,d_pathCapacities,d_flowMaxtrix,d_pathCapacities);
      if (tempCapacity == 0){
        break;
      }
      flow += tempCapacity;
      int v = t;

      //copy from host(my computer) to device(GPU)
      hipMemcpy(d_flowMaxtrix,flowMatrix,g->n * g->n * sizeof(int),hipMemcpyHostToDevice);
      hipMemcpy(d_parents,parents,g->n * sizeof(int),hipMemcpyHostToDevice);
      // backtrack

      backTrack<<<1,1,>>>(d_parents,d_flowMaxtrix,s,v,tempCapacity,g_>n);
      //copy device to host
      hipMemcpy(flowMatrix,d_flowMaxtrix,g->n * g->n * sizeof(int),hipMemcpyDeviceToHost);
      hipMemcpy(parents,d_parents,g->n * sizeof(int),hipMemcpyDeviceToHost);
    }
    Flow *result = (Flow *)malloc(sizeof(Flow));
    result->maxFlow = flow;
    result->finalEdgeFlows = flowMatrix;
    free(parents);
    free(pathCapacities);

    hipFree(d_flowMaxtrix);
    hipFree(d_parents);
    hipFree(d_pathCapacities);
    return result;


Flow *dinicGpu(Graph *g, int s, int t){
  // TODO: implement this
}