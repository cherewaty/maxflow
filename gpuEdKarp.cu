#include "hip/hip_runtime.h"
#include "gpuEdKarp.h"
#include <algorithm>
#include <limits>
#include <queue>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define IDX(i, j, n) ((i) * (n) + (j))
/*
__device__ int indexFinder(int i,int n, int j){
    return ((i) * (n) + (j));
}

__global__ void backTrack(int *parents,int *flowMatrix, int s,int v,int tempCapacity,int n){
    while (v != s){
        int u = parents[v];
        flowMatrix[indexFinder(u,v,n)] += tempCapacity;
        flowMatrix[indexFinder(v,u,n)] -= tempCapacity;
        v = u;
      }
}

__global__ void nextQueue(int queueSize,int *addedToQueue,bool *hasResult,int *result,int *queue,int *queueIndex int *parents, int *pathCapacities,int u,int n,int *capacities, int *flowMatrix){
    *addedToQueue = 0;
    for (int v = 0; v < n; v++){
        if (u == v){
          continue;
        }
        int residual = capacities[indexFinder(u, v, n)] - flowMatrix[indexFinder(u, v, n)];
        if ((residual > 0) && (parents[v] == -1)){
          parents[v] = u;
          pathCapacities[v] = fminf(pathCapacities[u], residual);
          if (v != t){
            queue[queueSize + *addedToQueue]=v;
            *addedToQueue++;
          }else{
            *hasResult = true;
            *result = pathCapacities[t];
            break;
          }
        }
      }
}

//Code Reference https://github.com/kaletap/bfs-cuda-gpu/blob/master/src/gpu/simple/bfs_simple.cu
int BFS(Graph *g, int *flowMatrix, int *parents, int *pathCapacities, int s, int t,int *d_parents,int *d_flowMaxtrix,int *d_pathCapacities){
  memset(parents, -1, (g->n * sizeof(int)));
  memset(pathCapacities, 0, (g->n * sizeof(int)));
  parents[s] = s;
  pathCapacities[s] = std::numeric_limits<int>::max();

  int currentQueueSize = 1;

  int *queue;
  int queueIndex = 0;
  int addedToQueue = 0;
  int result = 0;
  
  int *d_queue;
  int *d_queueIndex;
  int *d_addedToQueue;
  int *d_result;
  bool *d_hasResult;
  int size = g->n * sizeof(int);

  malloc(queue,size);
  hipMalloc((void **)&d_queue, size);
  hipMalloc((void**)&d_queueIndex,sizeof(int));
  hipMalloc((void**)&d_addedToQueue,sizeof(int));
  hipMalloc((void**)&d_hasResult,sizeof(bool));
  hipMalloc((void**)&d_result,sizeof(int));
  *hasResult = false;
  queue[0] = s;

  while(currentQueueSize > 0){
    int u = queue[queueIndex];
    hipMemcpy(d_hasResult,hasResult,sizeof(bool),hipMemcpyHostToDevice);
    hipMemcpy(d_resutl,result,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_addedToQueue,addedToQueue,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_queue,queue,size,hipMemcpyHostToDevice);
    hipMemcpy(d_queueIndex,queueIndex,sizeof(int),hipMemcpyHostToDevice);

    nextQueue<<<1,1>>>(currentQueueSize,d_addedToQueue,d_hasResult,d_result,d_queue,d_queueIndex,d_parents,d_pathCapacities,u,g->n,d_capacities,d_flowMaxtrix);
    hipMemcpy(hasResult,d_hasResult,sizeof(bool),hipMemcpyDeviceToHost);
    hipMemcpy(result,d_resutl,sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(addedToQueue,d_addedToQueue,sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(parents,d_parents,size,hipMemcpyDeviceToHost);
    hipMemcpy(pathCapacities,d_pathCapacities,size,hipMemcpyDeviceToHost);
    if(hasResult){
        free(queue)
        hipFree(d_hasResult);
        hipFree(d_result);
        hipFree(d_queueIndex);
        hipFree(d_addedToQueue);
        hipFree(d_queue);
        return result
    }else {
        hipMemcpy(queue,d_queue,size,hipMemcpyDeviceToHost);
        currentQueueSize = addedToQueue -1;
        queueIndex++;
    }

  }
  free(queue)
  hipFree(d_hasResult);
  hipFree(d_result);
  hipFree(d_queueIndex);
  hipFree(d_addedToQueue);
  hipFree(d_queue);
  return 0;
}

Flow *edKarpGpu(Graph *g, int s, int t){
    int flow = 0;
    int *flowMatrix = (int *)malloc(g->n * g->n * sizeof(int));
    int *parents = (int *)malloc(g->n * sizeof(int));
    int *pathCapacities = (int *)malloc(g->n* sizeof(int));
    
    
    int *d_flowMaxtrix;
    int *d_parents;
    int *d_pathCapacities;

    hipMalloc((void **)&d_flowMaxtrix,g->n * g->n * sizeof(int));
    hipMalloc((void **)&d_parents,g->n * sizeof(int));
    hipMalloc((void **)&d_pathCapacities,g->n* sizeof(int));

    while (true){
      int tempCapacity = BFS(g, flowMatrix, parents, pathCapacities, s, t,d_pathCapacities,d_flowMaxtrix,d_pathCapacities);
      if (tempCapacity == 0){
        break;
      }
      flow += tempCapacity;
      int v = t;

      //copy from host(my computer) to device(GPU)
      hipMemcpy(d_flowMaxtrix,flowMatrix,g->n * g->n * sizeof(int),hipMemcpyHostToDevice);
      hipMemcpy(d_parents,parents,g->n * sizeof(int),hipMemcpyHostToDevice);
      // backtrack

      backTrack<<<1,1,>>>(d_parents,d_flowMaxtrix,s,v,tempCapacity,g_>n);
      //copy device to host
      hipMemcpy(flowMatrix,d_flowMaxtrix,g->n * g->n * sizeof(int),hipMemcpyDeviceToHost);
      hipMemcpy(parents,d_parents,g->n * sizeof(int),hipMemcpyDeviceToHost);
    }
    Flow *result = (Flow *)malloc(sizeof(Flow));
    result->maxFlow = flow;
    result->finalEdgeFlows = flowMatrix;
    free(parents);
    free(pathCapacities);

    hipFree(d_flowMaxtrix);
    hipFree(d_parents);
    hipFree(d_pathCapacities);
    return result;
}*/



/*
*   Source from https://github.com/vulq/Flo
*/

int BFS(Graph *g, int *flowMatrix, int *parents, int *pathCapacities, int s, int t)
{
  memset(parents, -1, (g->n * sizeof(int)));
  memset(pathCapacities, 0, (g->n * sizeof(int)));
  parents[s] = s;
  pathCapacities[s] = std::numeric_limits<int>::max();
  std::queue<int> bfsQueue;
  bfsQueue.push(s);
  while (!bfsQueue.empty())
  {
    int u = bfsQueue.front();
    bfsQueue.pop();
    for (int v = 0; v < g->n; v++)
    {
      if (u == v)
        continue;
      int residual = g->capacities[IDX(u, v, g->n)] - flowMatrix[IDX(u, v, g->n)];
      if ((residual > 0) && (parents[v] == -1))
      {
        parents[v] = u;
        pathCapacities[v] = std::min(pathCapacities[u], residual);
        if (v != t)
        {
          bfsQueue.push(v);
        }
        else
        {
          int result = pathCapacities[t];
          return result;
        }
      }
    }
  }
  return 0;
}

// Edmonds-Karp algorithm to find max s-t flow
Flow *edKarpSeq(Graph *g, int s, int t)
{
  int flow = 0;
  int *flowMatrix = (int *)calloc((g->n * g->n), sizeof(int));
  int *parents = (int *)malloc(g->n * sizeof(int));
  int *pathCapacities = (int *)calloc(g->n, sizeof(int));
  while (true)
  {
    int tempCapacity = BFS(g, flowMatrix, parents, pathCapacities, s, t);
    if (tempCapacity == 0)
    {
      break;
    }
    flow += tempCapacity;
    int v = t;
    // backtrack
    while (v != s)
    {
      int u = parents[v];
      flowMatrix[IDX(u, v, g->n)] += tempCapacity;
      flowMatrix[IDX(v, u, g->n)] -= tempCapacity;
      v = u;
    }
  }
  Flow *result = (Flow *)malloc(sizeof(Flow));
  result->maxFlow = flow;
  result->finalEdgeFlows = flowMatrix;
  free(parents);
  free(pathCapacities);
  return result;
}
